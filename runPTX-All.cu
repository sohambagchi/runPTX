#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <iostream>
#include <cstring>
#include <fstream>
#include <streambuf>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>

#define IMPORTED_KERNEL


#ifndef IMPORTED_KERNEL

__global__ void memoryTest(long long * starting_address, long long * before1_load, long long * after1_load, long long * duration1, long long * before2_load, long long * after2_load, long long * duration2) {

    // long long int before1 = 0;
    // long long int after1 = 0;

    *before1_load = 0;
    *after1_load = 0;
    // *duration1 = 0;
    *before2_load = 0;
    *after2_load = 0;
    // *duration2 = 0;


    long long * start1 = starting_address;
    long long * start2 = starting_address;
    unsigned int i = 0;

    for (i = 0; i < 2047; i++) {
        starting_address[i] = (long long) &starting_address[i + 1];
    }

    i = 0;

    asm volatile("mov.u64 %0, %clock64 ;" : "=l"(*before1_load));
    // *before1_load = clock64();

    for (i = 0; i < 2048; i++) {
        start1 = (long long *) *start1;
    }

    asm volatile("mov.u64 %0, %clock64 ;" : "=l"(*after1_load));
    // *after1_load = clock64();
    
    asm volatile("mov.u64 %0, %clock64 ;" : "=l"(*before2_load));
    // *before2_load = clock64();

    for (i = 0; i < 2048; i++) {
        start2 = (long long *) *start2;
    }

    // asm volatile("mov.u64 %0, %clock64 ;" : "=l"(*after_load));
    asm volatile("mov.u64 %0, %clock64 ;" : "=l"(*after2_load));
    // *after2_load = clock64();

    *duration1 = *after1_load - *before1_load;
    *duration2 = *after2_load - *before2_load;
}
#endif


#ifdef IMPORTED_KERNEL
#define SAFE(X) if ((err = X) != hipSuccess) printf("CUDA error %d at %d\n", (int)err, __LINE__)
#else
#define SAFE(x) if (0 != x) { abort(); }
#endif



#ifdef IMPORTED_KERNEL

hipDevice_t cuDevice;
hipCtx_t cuContext;
hipModule_t cuModule;
hipFunction_t memoryTest;
hipError_t err;

hipDeviceptr_t largeObjectArray;
hipDeviceptr_t * before;
hipDeviceptr_t * after;
hipDeviceptr_t * duration;

#else
long long * largeObjectArray;
long long * before1;
long long * after1;
long long * duration1;
long long * before2;
long long * after2;
long long * duration2;
#endif

int main(int argc, char * argv[]) {
    srand(time(NULL));
    
    int devID = 0;

    long long * _before;
    long long * _after;
    long long * _duration;

    hipEvent_t start, stop;
    float milliseconds; 

    hipEventCreate(&start);
    hipEventCreate(&stop);

    #ifdef IMPORTED_KERNEL
    SAFE(hipInit(0));
    SAFE(hipDeviceGet(&cuDevice, devID));

    SAFE(hipCtxCreate(&cuContext, 0, cuDevice));

    if (argc < 5) {
        printf("Usage: %s <num_requests> <cache_level> <memory_type> <ptx_filename>\n", argv[0]);
        printf("cache_level: L1, L2, BASE\n");
        printf("memory_type: CPU, GPU, UM\n");
        return 1;
    }

    int num_requests = atoi(argv[1]);
    char * cache_level = argv[2];
    char * memory_type = argv[3];
    char * ptx = argv[4];
  
    SAFE(hipModuleLoad(&cuModule, ptx));

    printf("Loaded module\n");

    char * fn_name = (char *) malloc(strlen("_Z10memoryTestPx") + (((num_requests * 3) + 1) * 2));

    snprintf(fn_name, strlen("_Z10memoryTestPx") + 1, "_Z10memoryTestPx");

    for (int i = 0; i < (num_requests * 3) + 1; i++) {
        snprintf(fn_name + strlen(fn_name), 3, "S_");
    }

    SAFE(hipModuleGetFunction(&memoryTest, cuModule, fn_name));

    printf("Got function\n");

    int region_size;
    int load_size;

    if (strcmp(cache_level, "L1") == 0) {
        region_size = 16384;
        load_size = 16384;
    } else if (strcmp(cache_level, "L2") == 0) {
        region_size = 786432;
        load_size = 786432;
    } else if (strcmp(cache_level, "BASE") == 0) {
        region_size = 6533595;
        load_size = 32768;
    } else {
        printf("Invalid region size\n");
        return 1;
    }

    if (strcmp(memory_type, "CPU") == 0) {
        SAFE(hipMemAllocHost((void **) &largeObjectArray, region_size * sizeof(long long)));
    } else if (strcmp(memory_type, "GPU") == 0) {
        SAFE(hipMalloc(&largeObjectArray, region_size * sizeof(long long)));
    } else if (strcmp(memory_type, "UM") == 0) {
        SAFE(hipMallocManaged(&largeObjectArray, region_size * sizeof(long long), hipMemAttachGlobal));
    } else {
        printf("Invalid memory type\n");
        return 1;
    }


    before = (hipDeviceptr_t *) malloc(sizeof(hipDeviceptr_t) * num_requests);
    after = (hipDeviceptr_t *) malloc(sizeof(hipDeviceptr_t) * num_requests);
    duration = (hipDeviceptr_t *) malloc(sizeof(hipDeviceptr_t) * num_requests);
    _before = (long long *) malloc(sizeof(long long) * num_requests);
    _after = (long long *) malloc(sizeof(long long) * num_requests);
    _duration = (long long *) malloc(sizeof(long long) * num_requests);

    for (int i = 0; i < num_requests; i++) {
        SAFE(hipMalloc(&before[i], sizeof(long long)));
        SAFE(hipMalloc(&after[i], sizeof(long long)));
        SAFE(hipMalloc(&duration[i], sizeof(long long)));
    }
    
    #else


    SAFE(hipMalloc(&largeObjectArray, region_size * sizeof(long long)));
    
    
    SAFE(hipMalloc(&before1, sizeof(long long)));
    SAFE(hipMalloc(&after1, sizeof(long long)));
    SAFE(hipMalloc(&duration1, sizeof(long long)));

    SAFE(hipMalloc(&before2, sizeof(long long)));
    SAFE(hipMalloc(&after2, sizeof(long long)));
    SAFE(hipMalloc(&duration2, sizeof(long long)));
    #endif

    printf("Done allocations\n");

    printf("Memory Type: %s | Cache Level: %s\n", memory_type, cache_level);

    #ifdef IMPORTED_KERNEL

    void ** args = (void **) malloc(sizeof(void *) * (num_requests * 3) + 1);
    args[0] = &largeObjectArray;

    int k = 1;
    for (int i = 0; i < num_requests; i++) {
        args[k++] = &before[i];
        args[k++] = &after[i];
        args[k++] = &duration[i];
    }

    // hipEventRecord(start);
    SAFE(hipModuleLaunchKernel(memoryTest, 1, 1, 1, 1, 1, 1, 0, NULL, args, NULL));
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    #else
    memoryTest<<<1,1>>>(largeObjectArray, before1, after1, duration1, before2, after2, duration2);
    #endif
    
    #ifdef IMPORTED_KERNEL
    SAFE(hipCtxSynchronize());
    #else
    SAFE(hipDeviceSynchronize());
    #endif

    #ifdef IMPORTED_KERNEL
    for (int i = 0; i < num_requests; i++) {
        SAFE(hipMemcpyDtoH(&_before[i], before[i], sizeof(long long)));
        SAFE(hipMemcpyDtoH(&_after[i], after[i], sizeof(long long)));
        SAFE(hipMemcpyDtoH(&_duration[i], duration[i], sizeof(long long)));
    }

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    milliseconds *= 1e6;
    #else
    SAFE(hipMemcpy(&_before1, before1, sizeof(int), hipMemcpyDeviceToHost));
    SAFE(hipMemcpy(&_after1, after1, sizeof(int), hipMemcpyDeviceToHost));
    SAFE(hipMemcpy(&_duration1, duration1, sizeof(int), hipMemcpyDeviceToHost));
    SAFE(hipMemcpy(&_before2, before2, sizeof(int), hipMemcpyDeviceToHost));
    SAFE(hipMemcpy(&_after2, after2, sizeof(int), hipMemcpyDeviceToHost));
    SAFE(hipMemcpy(&_duration2, duration2, sizeof(int), hipMemcpyDeviceToHost));
    #endif

    for (int i = 0; i < num_requests; i++) {
        printf("Before %d: %lld\n", i, _before[i]);
        printf("After %d: %lld\n", i, _after[i]);
        printf("Duration %d: %lld\n", i, _duration[i]);
        printf("Per-Load Duration %d (/%d): %f\n", i, load_size, (double)_duration[i] / (double) load_size);
    }

    // printf("Kernel Duration: %f | %f\n", milliseconds, milliseconds / (float) load_size);


    #ifdef IMPORTED_KERNEL
    
    if (strcmp(memory_type, "CPU") == 0) {
        SAFE(hipHostFree((void*) largeObjectArray));
    } else {
        SAFE(hipFree(largeObjectArray));
    }
    
    
    for (int i = 0; i < num_requests; i++) {
        SAFE(hipFree(before[i]));
        SAFE(hipFree(after[i]));
        SAFE(hipFree(duration[i]));
    }
    free(before);
    free(after);
    free(duration);

    free(_before);
    free(_after);
    free(_duration);

    free(args);
    #else
    SAFE(hipFree(largeObjectArray));
    SAFE(hipFree(before1));
    SAFE(hipFree(after1));
    SAFE(hipFree(duration1));

    SAFE(hipFree(before2));
    SAFE(hipFree(after2));
    SAFE(hipFree(duration2));
    #endif

}